#include "hip/hip_runtime.h"
/**
 * simple.cu
 * date:		2008.06
 * description:	 	fill an array with consecutive values	
 * notes:		compile with nvcc and parent code, simple.c:
 * 				"nvcc simple.c simple.cu"
 * This program is very similar to a program that appears on a CUDA tutorial
 * published in Dr. Dobbs Journal.
 * The tutorial is available at:
 * http://www.ddj.com/hpc-high-performance-computing/207200659
 * Andrew Bellenir's matrix multiplication was also used as a basis for this example
**/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cutil.h>
/* 
 * In CUDA it is necessary to define block sizes
 * The grid of data that will be worked on is divided into blocks
 */
#define BLOCK_SIZE 32
/**
 * This is the function that will be executed in each and every one
 * of the stream processors
 * The __global__ directive identifies this function as being
 * an executable kernel on the CUDA device.
 * All kernesl must be declared with a return type void 
 */ 
__global__ void cu_fillArray(int *array_d){
        int x;
	/* blockIdx.x is a built-in variable in CUDA
           that returns the blockId in the x axis
           of the block that is executing this block of code
           threadIdx.x is another built-in variable in CUDA
           that returns the threadId in the x axis
           of the thread that is being executed by this
           stream processor in this particular block
        */
	x=blockIdx.x*BLOCK_SIZE+threadIdx.x;
	array_d[x]=x;
}

/**
 * This function is called in the host computer.
 * It, in turn, calls the function that is executed on the GPU.
 * Recall that:
 *  The host computer and the GPU card have separate memories
 *  Hence it will be necessary to 
 *    - Allocate memory in the memory on the GPU 
 *    - Copy the variables that will be operated from the memory 
 *      in the host to the corresponding variable in the GPU memory
 *    - Describe the configuration of the grid and the block size
 *    - Call the kernel, the code that will be executed on the GPU
 *    - Once the kernel has finished executing, copy back
 *      the results from the memory of the GPU to the memory on the host
 */
extern "C" void fillArray(int *array,int arraySize){
	//a_d is the GPU counterpart of the array that exists on the host memory 
	int *array_d;
	hipError_t result;
	//allocate memory on device
	// hipMalloc allocates space in the memory of the GPU card
	result = hipMalloc((void**)&array_d,sizeof(int)*arraySize);
	if (result != hipSuccess) {
		printf("hipMalloc failed.");
		exit(1);
	}
	//copy the array into the variable array_d in the device 
	// The memory from the host is being copied to the corresponding variable
	// in the GPU global memory
	result = hipMemcpy(array_d,array ,sizeof(int)*arraySize,hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		printf("hipMemcpy failed.");
		exit(1);
	}

	//execution configuration...
	// Indicate the dimension of the block
	dim3 dimblock(BLOCK_SIZE);
	// Indicate the dimension of the grid measured in blocks
	dim3 dimgrid(arraySize/BLOCK_SIZE);
	//actual computation: Call the kernel, the function that is
	// executed by each and every stream processor on the GPU card
	cu_fillArray<<<dimgrid,dimblock>>>(array_d);
	//read results back:
	// Copy the results from the memory in the GPU back to the memory on the host
	result = hipMemcpy(array,array_d,sizeof(int)*arraySize,hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		printf("hipMemcpy failed.");
		exit(1);
	}
	// Release the memory on the GPU card
	result = hipFree(array_d);
	if (result != hipSuccess) {
		printf("hipFree failed.");
		exit(1);
	}
}

